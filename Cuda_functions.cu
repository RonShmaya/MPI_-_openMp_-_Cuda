#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "Cuda_functions.h"
#include "Input_Output_Manager.h"
#include "Data_Manager.h"
#include "Tools.h"

/*Calculate one object match in picture on specific index (x,y)*/
__device__ int is_match(BYTE* picture_elements, int picture_dim, int x, int y,BYTE* object_elements ,int object_dim, float matching){
	float delta = 0;
	float tmp;
	for (int row = 0; row < object_dim; ++row) {
		for (int col = 0; col < object_dim; ++col) {
									//[x]                      //[y]
			tmp = (picture_elements[ ((x+row)*picture_dim)  +  y+col] - object_elements[row*object_dim+col]) /(float)
																(picture_elements[ ((x+row)*picture_dim)  +  y+col]);
			delta+= tmp < 0 ? tmp * -1 : tmp; //(add ABS)
			if(delta > matching)
				return NOT_MATCH;
		}
	}
	
	return ITS_MATCH;
}
/*Calclate all the pictures match until 1024*1024 indexes
blockIdx.x = picture index
blockIdx.y = object index
threadIdx.x = picture row 
gridDim.y = num if objects
*/
__global__  void find_objects_in_pictures(Search_Data search_data,BYTE* picturesElements,int picturesElementsJumps, BYTE* objectsElements,int objectsElementsJumps,int* results) {

	int pic_dim = search_data.pictures[blockIdx.x].dimention;
	int obj_dim = search_data.objects[blockIdx.y].dimention;
	
	int num_of_searching_in_one_axis = pic_dim - obj_dim + 1;
	
	/*Calculate result index*/
	int result_picture_size = (gridDim.y)*blockDim.x;
	int result_object_size = blockDim.x;
	long result_index = (blockIdx.x*result_picture_size)   +   (blockIdx.y*result_object_size)   +threadIdx.x;
	results[result_index] = 0;
	
	if(obj_dim  > pic_dim)
		return;
	if(threadIdx.x >=  num_of_searching_in_one_axis)
		return;

	int res = 0;
	for (int i = 0; i < num_of_searching_in_one_axis; ++i) { 
		res = is_match(
				picturesElements + (blockIdx.x * picturesElementsJumps),
				pic_dim, 
				threadIdx.x ,
				i,
				objectsElements + (blockIdx.y * objectsElementsJumps),
				obj_dim,
				search_data.matching);

			if(res == ITS_MATCH ){ //every match is x y and object id -> populate into one integer
				results[result_index] = blockIdx.y << ID_SHIFT;
				results[result_index] |= threadIdx.x << X_SHIFT;
				results[result_index] |= i << Y_SHIFT;
				results[result_index] |=  ITS_MATCH;
				return;
			}
	}

}

hipError_t err = hipSuccess;

/*Calclate all the pictures match until 1024*1024 indexes*/
int* find_matches_by_cuda(Search_Data* h_search_data,int picture_start ,int picture_end){

    
	Search_Data to_cuda_search_data;
	to_cuda_search_data.matching = h_search_data->matching;
	BYTE* host_pictures_elements; /*all the pictures elements*/
	BYTE* host_objects_elements; /*all the objects elements*/
	
	int picture_max_size = (get_picture_max_dim() * get_picture_max_dim());
	int object_max_size = (get_object_max_dim() * get_object_max_dim());
	long pictures_elements_size = ((picture_end - picture_start) * picture_max_size);
	long objects_elements_size = (h_search_data->num_of_objects * object_max_size);
	long result_size = (picture_end - picture_start) * h_search_data->num_of_objects * get_picture_max_dim();
	
	host_pictures_elements = (BYTE*) malloc(sizeof(BYTE)*pictures_elements_size);
	if(verify_allocate_memory_succeded(host_pictures_elements) == ALLOCATE_MEMORY_FAILED) exit(0);


	/*fill all the pictures elements*/
	for (int picture = picture_start; picture < picture_end; picture++) {
		int pic_dim = h_search_data->pictures[picture].dimention;
		memcpy(host_pictures_elements + ((picture-picture_start) * picture_max_size), h_search_data->pictures[picture].elements, sizeof(BYTE)*pic_dim*pic_dim);
	} 

	host_objects_elements = (BYTE*) malloc(sizeof(BYTE)*objects_elements_size);
	if(verify_allocate_memory_succeded(host_objects_elements) == ALLOCATE_MEMORY_FAILED) exit(0);


	/*fill all the objects elements*/
	for (int object = 0; object < h_search_data->num_of_objects; object++) {
		int obj_dim = h_search_data->objects[object].dimention;
		memcpy(host_objects_elements + (object * object_max_size), h_search_data->objects[object].elements, sizeof(BYTE)*obj_dim*obj_dim);
	}


	BYTE* to_cuda_pictures_elements;
	BYTE* to_cuda_objects_elements;
	
	err = hipMalloc((void **)&(to_cuda_pictures_elements), sizeof(BYTE)*pictures_elements_size);
	verify_cuda_succedded(__LINE__);

	// Copy data from host to the GPU memory
	err = hipMemcpy(to_cuda_pictures_elements, host_pictures_elements, sizeof(BYTE)*pictures_elements_size, hipMemcpyHostToDevice);
	verify_cuda_succedded(__LINE__);


	err = hipMalloc((void **)&(to_cuda_objects_elements), sizeof(BYTE)*objects_elements_size);
	verify_cuda_succedded(__LINE__);

	// Copy data from host to the GPU memory
	err = hipMemcpy(to_cuda_objects_elements, host_objects_elements, sizeof(BYTE)*objects_elements_size, hipMemcpyHostToDevice);
	verify_cuda_succedded(__LINE__);


	// Will contain all the results
	int* to_cuda_results;
	err = hipMalloc((void **)&(to_cuda_results), sizeof(int)*(result_size));
	verify_cuda_succedded(__LINE__);


	err = hipMalloc((void **)&(to_cuda_search_data.pictures), sizeof(Picture) * (picture_end - picture_start));
	verify_cuda_succedded(__LINE__);

	// Copy data from host to the GPU memory
	err = hipMemcpy(to_cuda_search_data.pictures, h_search_data->pictures + picture_start, sizeof(Picture) * (picture_end - picture_start), hipMemcpyHostToDevice);
	verify_cuda_succedded(__LINE__);


	err = hipMalloc((void **)&(to_cuda_search_data.objects), sizeof(Picture) * (h_search_data->num_of_objects));
	verify_cuda_succedded(__LINE__);

	// Copy data from host to the GPU memory
	err = hipMemcpy(to_cuda_search_data.objects, h_search_data->objects, sizeof(Picture) * (h_search_data->num_of_objects), hipMemcpyHostToDevice);
	verify_cuda_succedded(__LINE__);
	
	
	dim3 DimGrid((picture_end - picture_start),h_search_data->num_of_objects);
	if(get_picture_max_dim() > CUDA_MAX_THREAD){
		printf("\n----------- PICTURE MAX SIZE is the cuda dim blocks and it's > %d, cut the the picture to avoid Cuda error\n",CUDA_MAX_THREAD);
		dim3 DimBlock(CUDA_MAX_THREAD);
		find_objects_in_pictures<<<DimGrid, DimBlock>>>(to_cuda_search_data,to_cuda_pictures_elements,picture_max_size,to_cuda_objects_elements,object_max_size,to_cuda_results);
	}
	else{
		dim3 DimBlock(get_picture_max_dim());
		find_objects_in_pictures<<<DimGrid, DimBlock>>>(to_cuda_search_data,to_cuda_pictures_elements,picture_max_size,to_cuda_objects_elements,object_max_size,to_cuda_results);
	}
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf( "Failed in cuda action -  %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	
	int* results = (int*) malloc(sizeof(int)*(result_size));
	if(verify_allocate_memory_succeded(results) == ALLOCATE_MEMORY_FAILED) exit(0);
	
	// Copy data from GPU memory to host
	err = hipMemcpy(results, to_cuda_results, sizeof(int)*(result_size), hipMemcpyDeviceToHost);
	verify_cuda_succedded(__LINE__);

	//Free all cuda allocate
	hipDeviceReset();


	free(host_pictures_elements);
	free(host_objects_elements);

	return results;

}
/*verify Cuda err*/
void verify_cuda_succedded(int line){
	if (err != hipSuccess) {
		printf("Failed in cuda action - %s in Line %d\n", hipGetErrorString(err),line);
		exit(0);
	}
}
